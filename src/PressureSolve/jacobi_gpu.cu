#include "hip/hip_runtime.h"

#include <cstdint>
#include <SwirlCraft/grid.h>
#include <SwirlCraft/PressureSolve/jacobi_gpu.h>

namespace SwirlCraft
{
    template <typename T>
    __global__ void jacobiIterationKernel(T* f, T* f_old, const T* g, const T* collision, const T c0, const T* c, const int64_t* strides, const size_t n, uint32_t Dims)
    {
        const auto idx = blockDim.x * blockIdx.x + threadIdx.x;

        if (idx < n && collision[idx] > 0)
        {
            T A = 0;
            for (uint32_t j = 0; j < Dims; j++)
            {
                const auto stride = strides[j];
                const T a1 = collision[idx-stride] > 0 ? f_old[idx-stride] : f_old[idx];
                const T a2 = collision[idx+stride] > 0 ? f_old[idx+stride] : f_old[idx];
                A += c[j] * (a1 + a2);
            }
            f[idx] = A - c0 * g[idx];
        }
    }
    
    
    template <typename T, uint32_t Dims>
    void jacobiSolve_gpu(T* f_d, T* f_old_d, const T* g_d, const T* collision_d, const Grid<T, Dims>& grid, const int32_t maxIterations, const int32_t blockSize)
    {
        T dxn2[Dims];
        T c[Dims];
        T sum = 0;
        const size_t N = grid.N;
        for (size_t i = 0; i < Dims; i++)
        {
            const T dx = grid.dx[i];
            dxn2[i] = 1 / (dx*dx);
            sum += dxn2[i];
        }

        int numBlocks = static_cast<int>(ceil(static_cast<double>(grid.N) / blockSize));

        const T c0 = static_cast<T>(0.5) / sum;
        for (size_t i = 0; i < Dims; i++)
        {
            c[i] = c0 * dxn2[i];
        }

        T* c_d;
        int64_t* strides_d;
        hipMalloc(&c_d, Dims * sizeof(T));
        hipMalloc(&strides_d, Dims * sizeof(int64_t));
        hipMemcpy(c_d, c, Dims * sizeof(T), hipMemcpyHostToDevice);
        hipMemcpy(strides_d, grid.stride, Dims * sizeof(int64_t), hipMemcpyHostToDevice);

        for (int32_t iter = 0; iter < maxIterations; iter++)
        {
            hipMemcpy(f_old_d, f_d, grid.N * sizeof(T), hipMemcpyDeviceToDevice);
            jacobiIterationKernel<<<numBlocks, blockSize>>>(f_d, f_old_d, g_d, collision_d, c0, c_d, strides_d, N, Dims);
        }

        hipFree(c_d);
        hipFree(strides_d);
    }
    
}


template void SwirlCraft::jacobiSolve_gpu(float*, float*, const float*, const float*, const SwirlCraft::Grid<float, 2u>&, const int32_t, const int32_t);
template void SwirlCraft::jacobiSolve_gpu(float*, float*, const float*, const float*, const SwirlCraft::Grid<float, 3u>&, const int32_t, const int32_t);
template void SwirlCraft::jacobiSolve_gpu(double*, double*, const double*, const double*, const SwirlCraft::Grid<double, 2u>&, const int32_t, const int32_t);
template void SwirlCraft::jacobiSolve_gpu(double*, double*, const double*, const double*, const SwirlCraft::Grid<double, 3u>&, const int32_t, const int32_t);
